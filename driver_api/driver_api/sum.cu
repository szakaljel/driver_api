#include "hip/hip_runtime.h"

// Vector addition (device code)

#include "sum.h"

extern "C" __global__ void Sum(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}